// includes, system
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <string>
#include <random>
#include <chrono>
#include <algorithm>

// includes, cuda
#include "hip/hip_runtime.h"


////////////////////////////////////////////////////////////////////////////////
#define BG_BBLUE_FG_BLACK "\033[104;30m"
#define BG_BLUE_FG_BLACK "\033[44;30m"
#define BG_BLUE_FG_WHITE "\033[44;37m"
#define BG_BLACK_FG_WHITE "\033[0m"
#define BG_WHITE_FG_BLACK "\033[30;107m"

// 0 - 0000 = empty
// 4 - 0100 = black man
// 5 - 0101 = black king
// 6 - 0110 = white man
// 7 - 0111 = white king
//
// 8 - 1000 in (tile_idx = 0) is used to save turn flag (1 - white, 0 - black)
//
// 8 tiles saved in one unsigned int with encoding as above
// example: 0100 0100 0100 0100 0000 0000 0000 0000
// indexing: 7 6 5 4 3 2 1 0

//////////////////////////////////////////////////////////////////////////////// - board state macros
#define SET_VAL_BOARD(idx, val, board) board[idx >> 3] ^= (board[idx >> 3] ^ val << ((idx & 7) << 2)) & (15 << ((idx & 7) << 2))
#define GET_VAL_BOARD(idx, board) board[idx >> 3] << 28 - ((idx & 7) << 2) >> 28
#define GET_VAL_BOARD_S(idx, board) idx > 31 ? 8 : board[idx >> 3] << 28 - ((idx & 7) << 2) >> 28
//#define IS_EMPTY(tile) (bool)(!tile)
#define IS_PIECE(tile) (bool)(tile & 4)
#define IS_WHITE(tile) (bool)(tile & 2)
#define IS_BLACK(tile) (bool)(~tile & 2)
#define IS_KING(tile) (bool)(tile & 1)
#define FLIP_TURN_FLAG(board) board[0] ^= 8
#define GET_TURN_FLAG(board) (bool)(board[0] & 8)
//////////////////////////////////////////////////////////////////////////////// - move_pos array macros
#define GET_BEATING_POS_FLAG(move_pos) (bool)(move_pos[3] & 1)
#define SET_BEATING_POS_FLAG(move_pos) move_pos[3] |= 1
#define GET_MOVE_CHECK_GUARD(move_pos) (bool)(move_pos[3] & 2)
#define SET_MOVE_CHECK_GUARD(move_pos) move_pos[3] |= 2
#define CLEAR_MOVE_CHECK_GUARD(move_pos) move_pos[3] &= ~2
#define GET_NUM_OF_MOVES(move_pos) move_pos[3] >> 2
#define SET_NUM_OF_MOVES(move_pos, num_of_moves) move_pos[3] |= num_of_moves << 2
#define GET_VAL_MOVE_POS(idx, move_pos) move_pos[idx >> 2] << 24 - ((idx & 3) << 3) >> 24
#define SET_VAL_MOVE_POS(idx, val, move_pos) move_pos[idx >> 2] |= val << ((idx & 3) << 3)
#define GET_PIECE_DIR_FLAG(dir, move_pos) (bool)((move_pos[2] << 30 - (dir << 1) >> 30) & 1)
#define SET_PIECE_DIR_FLAG(dir, move_pos) move_pos[2] |= 1 << (dir << 1)
#define GET_PIECE_BEATING_FLAG(dir, move_pos) (bool)((move_pos[2] << 30 - (dir << 1) >> 30) & 2)
#define SET_PIECE_BEATING_FLAG(dir, move_pos) move_pos[2] |= 2 << (dir << 1)
////////////////////////////////////////////////////////////////////////////////
void init_board(unsigned int board[4]);
void draw_board(unsigned int board[4]);
unsigned int get_left_upper_idx(unsigned int& cur_tile_idx);
unsigned int get_right_upper_idx(unsigned int& cur_tile_idx);
unsigned int get_left_lower_idx(unsigned int& cur_tile_idx);
unsigned int get_right_lower_idx(unsigned int& cur_tile_idx);
void get_move_possibility_loop_fun(unsigned int board[4], unsigned int move_pos[4], unsigned int& cur_idx, unsigned int& moves_idx);
void get_move_possibility(unsigned int board[4], unsigned int move_pos[4]);
////////////////////////////////////////////////////////////////////////////////
void get_piece_move_pos(unsigned int board[4], unsigned int move_pos[4], unsigned int& idx);
void move_piece(unsigned int board[4], unsigned int& cur_tile_idx, unsigned int (*get_dir_idx_ptr)(unsigned int&));
////////////////////////////////////////////////////////////////////////////////
void game_loop(unsigned int board[4], void (*white_player)(unsigned int*, unsigned int*), void (*black_player)(unsigned int*, unsigned int*));
void human_player(unsigned int board[4], unsigned int move_pos[4]);
void random_player(unsigned int board[4], unsigned int move_pos[4]);
unsigned int simulate_game(unsigned int board[4]);
unsigned int count_beating_sequences_for_piece(unsigned int board[4], unsigned int cur_tile_idx, unsigned int dir);
void MTS_CPU_player(unsigned int board[4]);
////////////////////////////////////////////////////////////////////////////////
void disp_moveable_pieces(unsigned int board[4], unsigned int move_pos[4]);
void disp_possible_dirs(unsigned int board[4], unsigned int move_pos[4], unsigned int& idx);
void get_cords_from_console(char cords[2]);
unsigned int translate_cords_to_idx(const char cords[2]);
void translate_idx_to_cords(unsigned int idx, char cords[2]);
void get_end_state(unsigned int board[4]);
void disp_end_state(unsigned int* board);
////////////////////////////////////////////////////////////////////////////////
void testing_function();
void test_get_idx_funs(unsigned int board[4]);
void test_get_move_possibility(unsigned int board[4], unsigned int move_pos[4]);
void test_get_move_possibility_board_init(unsigned int board[4], unsigned int test_choice);
void test_get_move_possibility_init_loop(unsigned int board[4], int lower_bound = 1, int upper_bound = 7);
void test_get_piece_move_pos(unsigned int board[4], unsigned int move_pos[4], unsigned int idx);
void test_translate_cords_to_idx();
void test_translate_idx_to_cords();
//void bench(unsigned int board[4]);
////////////////////////////////////////////////////////////////////////////////

void init_board(unsigned int board[4])
{
    // white bottom
    board[0] = 1145324612; //1st 2nd rows
    board[1] = 17476; //3rd 4th rows
    board[2] = 1717960704; //5th 6th rows
    board[3] = 1717986918; //7th 8th rows
}

void draw_board(unsigned int board[4])
{
    unsigned short i = 0, left_side_idx = 1;
    bool white_first = true;

    std::cout << BG_BBLUE_FG_BLACK << "   ";
    for (char c = 'A'; c != 'I'; ++c)
        std::cout << ' ' << c << ' ';
    std::cout << BG_BLACK_FG_WHITE << std::endl;

    for (; i < 4; ++i) // i = board_idx
    {
        for (unsigned int j = 0; j < 8; ++j) // j = tile_in_board_idx
        {
            unsigned int tile = board[i] << (28 - (j << 2)) >> 28;
            
            if (j == 0 || j == 4) std::cout << BG_BBLUE_FG_BLACK << ' ' << left_side_idx++ << ' ';

            if (white_first) std::cout << BG_BBLUE_FG_BLACK << "   ";

            if (IS_PIECE(tile))
            {
                if (IS_WHITE(tile)) std::cout << BG_BLUE_FG_WHITE;
                else std::cout << BG_BLUE_FG_BLACK;
                if (IS_KING(tile)) std::cout << " K ";
                else std::cout << " @ ";
            }
            else std::cout << BG_BLUE_FG_BLACK << "   ";
            
            if (!white_first) std::cout << BG_BBLUE_FG_BLACK << "   ";

            if ((j & 3) == 3) // swap colors for second row
            {
                std::cout << BG_BLACK_FG_WHITE << std::endl;
                white_first = !white_first;
            }
        }
    }
}

////////////////////////////////////////////////////////////////////////////////

unsigned int get_left_upper_idx(unsigned int& cur_tile_idx)
{
    if (cur_tile_idx > 31 || !(cur_tile_idx >> 2)) return 32; // second condition is top row
    if (cur_tile_idx & 4) // even row (counting from 1)
    {
        if (cur_tile_idx & 3) // if not left-most
            return cur_tile_idx - 5;
        return 32;
    }
    else // odd row
    {
        return cur_tile_idx - 4;
    }
}

unsigned int get_right_upper_idx(unsigned int& cur_tile_idx)
{
    if (cur_tile_idx > 31 || !(cur_tile_idx >> 2)) return 32; // second cond chcks if top row
    if (cur_tile_idx & 4) // even row (counting from 1)
    {
        return cur_tile_idx - 4;
    }
    else // odd row
    {
        if (~cur_tile_idx & 3) // if not right-most
            return cur_tile_idx - 3;
        return 32;
    }
}

unsigned int get_left_lower_idx(unsigned int& cur_tile_idx)
{
    if (cur_tile_idx > 31 || (cur_tile_idx >> 2) == 7) return 32; // second cond chcks if bottom row
    if (cur_tile_idx & 4) // even row (counting from 1)
    {
        if (cur_tile_idx & 3) // if not left-most
            return cur_tile_idx + 3;
        return 32;
    }
    else // odd row
    {
        return cur_tile_idx + 4;
    }
}

unsigned int get_right_lower_idx(unsigned int& cur_tile_idx)
{
    if(cur_tile_idx > 31 || (cur_tile_idx >> 2) == 7) return 32; // second cond chcks if bottom row
    if (cur_tile_idx & 4) // even row (counting from 1)
    {
        return cur_tile_idx + 4;
    }
    else // odd row
    {
        if (~cur_tile_idx & 3) // if not right-most
            return cur_tile_idx + 5;
        return 32;
    }
}

void get_move_possibility_loop_fun(unsigned int board[4], unsigned int move_pos[4], unsigned int& cur_idx, unsigned int& moves_idx)
{
    unsigned int tile, tmp_idx, result;
    tile = GET_VAL_BOARD(cur_idx, board);
    if (IS_PIECE(tile) && (GET_TURN_FLAG(board) == IS_WHITE(tile)))
    {
        unsigned int (*get_dir_idx_ptr)(unsigned int&);
        for (unsigned int direction = 0; direction < 4; ++direction)
        {
            if (GET_TURN_FLAG(board) == (bool)(direction & 2) && !IS_KING(tile)) // do not check backwards movement
                continue;
            switch (direction)
            {
            case 0:
                get_dir_idx_ptr = &get_left_upper_idx;
                break;
            case 1:
                get_dir_idx_ptr = &get_right_upper_idx;
                break;
            case 2:
                get_dir_idx_ptr = &get_left_lower_idx;
                break;
            case 3:
                get_dir_idx_ptr = &get_right_lower_idx;
                break;
            default: return;
            }
            tmp_idx = get_dir_idx_ptr(cur_idx);
            if (tmp_idx == 32) continue;
            result = GET_VAL_BOARD(tmp_idx, board);
            if (GET_TURN_FLAG(board) != IS_WHITE(result) && IS_PIECE(result))
            {
                tmp_idx = get_dir_idx_ptr(tmp_idx);
                if (tmp_idx == 32) continue;
                result = GET_VAL_BOARD(tmp_idx, board);
                if (!IS_PIECE(result))
                {
                    if (!GET_BEATING_POS_FLAG(move_pos)) 
                    {
                        moves_idx = 0;
                        move_pos[0] = move_pos[1] = move_pos[2] = move_pos[3] = 0;
                        SET_BEATING_POS_FLAG(move_pos);
                    }
                    SET_VAL_MOVE_POS(moves_idx, cur_idx, move_pos);
                    ++moves_idx;
                    CLEAR_MOVE_CHECK_GUARD(move_pos);
                    return;
                }
            }
            else if (!IS_PIECE(result) && !GET_BEATING_POS_FLAG(move_pos) && !GET_MOVE_CHECK_GUARD(move_pos))
            {
                SET_VAL_MOVE_POS(moves_idx, cur_idx, move_pos);
                ++moves_idx;
                SET_MOVE_CHECK_GUARD(move_pos);
                continue;
            }
        }
        CLEAR_MOVE_CHECK_GUARD(move_pos);
    }
}

// Index of tile that can be moved is stored similarly as board representation, but in 8 bits instead of 4 bits
// Additionally some space in move_pos[2] is used for flags and saving number of indexes in the whole array
void get_move_possibility(unsigned int board[4], unsigned int move_pos[4])
{
    unsigned int moves_idx = 0;
    move_pos[0] = move_pos[1] = move_pos[2] = move_pos[3] = 0;
    for (unsigned int i = 0; i < 32; ++i)
        get_move_possibility_loop_fun(board, move_pos, i, moves_idx);
    SET_NUM_OF_MOVES(move_pos, moves_idx); // record number of possible moves
}

////////////////////////////////////////////////////////////////////////////////

// Index of tile that can be moved is stored similarly as board representation, but in 8 bits instead of 2 bits
// move_pos[2] is used for storing, the same spots as in get_move_possibility are used for beating available flag and number of indexes saved
void get_piece_move_pos(unsigned int board[4], unsigned int move_pos[4], unsigned int& idx)
{
    unsigned int tile, tmp_idx, result, move_counter = 0;
    move_pos[2] = move_pos[3] = 0;
    
    tile = GET_VAL_BOARD_S(idx, board);
    if (IS_PIECE(tile))
    {
        unsigned int (*get_dir_idx_ptr)(unsigned int&);
        for (unsigned int direction = 0; direction < 4; ++direction)
        {
            if (IS_WHITE(tile) == (bool)(direction & 2) && !IS_KING(tile)) // do not check backwards movement
                continue;
            switch (direction)
            {
            case 0:
                get_dir_idx_ptr = &get_left_upper_idx;
                break;
            case 1:
                get_dir_idx_ptr = &get_right_upper_idx;
                break;
            case 2:
                get_dir_idx_ptr = &get_left_lower_idx;
                break;
            case 3:
                get_dir_idx_ptr = &get_right_lower_idx;
                break;
            default: return;
            }
            tmp_idx = get_dir_idx_ptr(idx);
            if (tmp_idx == 32) continue;
            result = GET_VAL_BOARD(tmp_idx, board);
            if (IS_WHITE(tile) != IS_WHITE(result) && IS_PIECE(result)) // IS_PIECE = out of bounds guard
            {
                tmp_idx = get_dir_idx_ptr(tmp_idx);
                if (tmp_idx == 32) continue;
                result = GET_VAL_BOARD(tmp_idx, board);
                if (!IS_PIECE(result))
                {
                    if (!GET_BEATING_POS_FLAG(move_pos)) {
                        move_counter = 0;
                        SET_BEATING_POS_FLAG(move_pos);
                    }
                    SET_PIECE_BEATING_FLAG(direction, move_pos);
                    ++move_counter;
                }
            }
            else if (!IS_PIECE(result) && !GET_BEATING_POS_FLAG(move_pos))
            {
                SET_PIECE_DIR_FLAG(direction, move_pos);
                ++move_counter;
            }
        }
    }
    SET_NUM_OF_MOVES(move_pos, move_counter);
}

void move_piece(unsigned int board[4], unsigned int& cur_tile_idx, unsigned int (*get_dir_idx_ptr)(unsigned int&))
{
    if (cur_tile_idx > 31) return;
    
    unsigned int other_tile_idx = get_dir_idx_ptr(cur_tile_idx);
    if (other_tile_idx == 32) return;
    
    unsigned int cur_tile = GET_VAL_BOARD(cur_tile_idx, board);
    if (!(GET_VAL_BOARD(other_tile_idx, board)))
    {
        SET_VAL_BOARD(other_tile_idx, cur_tile, board);
        SET_VAL_BOARD(cur_tile_idx, 0, board);
    }
    else
    {
        SET_VAL_BOARD(other_tile_idx, 0, board);
        SET_VAL_BOARD(cur_tile_idx, 0, board);
        other_tile_idx = get_dir_idx_ptr(other_tile_idx);
        SET_VAL_BOARD(other_tile_idx, cur_tile, board);
    }
    if ((!IS_KING(cur_tile)) && ((IS_WHITE(cur_tile) && other_tile_idx < 4) || (IS_BLACK(cur_tile) && other_tile_idx > 27)))
        SET_VAL_BOARD(other_tile_idx, (cur_tile | 1), board); // promote to king
}

////////////////////////////////////////////////////////////////////////////////

void game_loop(unsigned int board[4], void (*white_player)(unsigned int*, unsigned int*), void (*black_player)(unsigned int*, unsigned int*))
{
    unsigned int move_pos[4];
    bool game_over = false;

    while (!game_over) // main loop
    {
        system("CLS");
        draw_board(board);
        std::cout << std::endl << (GET_TURN_FLAG(board) ? BG_WHITE_FG_BLACK : BG_BLACK_FG_WHITE) << (GET_TURN_FLAG(board) ? "White" : "Black") << "'s turn!" << BG_BLACK_FG_WHITE << std::endl << std::endl;
        //get_move_possibility(board, move_pos);
        //system("pause");

        if (GET_TURN_FLAG(board))
            white_player(board, move_pos);
        else
            black_player(board, move_pos);

        get_move_possibility(board, move_pos);
        if (0 == (GET_NUM_OF_MOVES(move_pos))) game_over = true; // end game if noone can move
    }
}

void human_player(unsigned int board[4], unsigned int move_pos[4])
{
    unsigned int choosen_idx1, choosen_idx2, dir;
    char cords[2];
    bool board_beating_flag, beating_sequence_in_progress = false;

    auto redraw_beginning = [board]()
    {
        system("CLS");
        draw_board(board);
        std::cout << std::endl << (GET_TURN_FLAG(board) ? BG_WHITE_FG_BLACK : BG_BLACK_FG_WHITE) << (GET_TURN_FLAG(board) ? "White" : "Black") << "'s turn!" << BG_BLACK_FG_WHITE << std::endl << std::endl;
    };
    auto redraw_first_stage = [board, move_pos, redraw_beginning]()
    {
        redraw_beginning();
        get_move_possibility(board, move_pos);
        disp_moveable_pieces(board, move_pos);
        std::cout << std::endl;
    };
    auto redraw_second_stage = [board, move_pos, &choosen_idx1, redraw_beginning]()
    {
        redraw_beginning();
        get_piece_move_pos(board, move_pos, choosen_idx1);
        disp_possible_dirs(board, move_pos, choosen_idx1);
        std::cout << std::endl;
    };

    human_player_reset:
    while (true) // piece choice loop
    {
        redraw_first_stage();
        get_cords_from_console(cords);
        choosen_idx1 = translate_cords_to_idx(cords);
        board_beating_flag = GET_BEATING_POS_FLAG(move_pos);

        get_piece_move_pos(board, move_pos, choosen_idx1);
        if (0 == (GET_NUM_OF_MOVES(move_pos)))
        {
            std::cout << std::endl << "This piece cannot move!" << std::endl << "Please choose a different piece!" << std::endl << std::endl;
            system("pause");
            continue;
        }
        else if (board_beating_flag != GET_BEATING_POS_FLAG(move_pos))
        {
            std::cout << std::endl << "BEATING POSSIBLE!" << std::endl << "Please choose a different piece!" << std::endl << std::endl;
            system("pause");
            continue;
        }
        break;
    }

    while (true) // move sequence loop
    {
        redraw_second_stage();
        get_cords_from_console(cords);
        choosen_idx2 = translate_cords_to_idx(cords);

        unsigned int (*get_dir_idx_ptr)(unsigned int&);
        for (dir = 0; dir < 4; ++dir)
        {
            if (dir < 2 && choosen_idx2 > choosen_idx1) continue;
            switch (dir)
            {
            case 0:
                get_dir_idx_ptr = &get_left_upper_idx;
                break;
            case 1:
                get_dir_idx_ptr = &get_right_upper_idx;
                break;
            case 2:
                get_dir_idx_ptr = &get_left_lower_idx;
                break;
            case 3:
                get_dir_idx_ptr = &get_right_lower_idx;
                break;
            default: goto human_player_reset;
            }
            if (choosen_idx2 != get_dir_idx_ptr(choosen_idx1)) continue;

            if (GET_BEATING_POS_FLAG(move_pos) && GET_PIECE_BEATING_FLAG(dir, move_pos))
            {
                board_beating_flag = IS_KING((GET_VAL_BOARD(choosen_idx1, board))); //memory recycling - dont mind the name
                move_piece(board, choosen_idx1, get_dir_idx_ptr);
                choosen_idx1 = get_dir_idx_ptr(choosen_idx2);
                if (board_beating_flag != (IS_KING((GET_VAL_BOARD(choosen_idx1, board)))))
                {
                    FLIP_TURN_FLAG(board);
                    return;
                }
                break;
            }
            else if (!GET_BEATING_POS_FLAG(move_pos) && GET_PIECE_DIR_FLAG(dir, move_pos))
            {
                move_piece(board, choosen_idx1, get_dir_idx_ptr);
                FLIP_TURN_FLAG(board);
                return;
            }
            std::cout << std::endl << "Impossible move!" << std::endl << "Please choose a different move!" << std::endl << std::endl;
            system("pause");
            if (beating_sequence_in_progress) break;
            goto human_player_reset; // reset move choice
        }
        if (dir == 4)
        {
            std::cout << std::endl << "Impossible move!" << std::endl << "Please choose a different move!" << std::endl << std::endl;
            system("pause");
            if (beating_sequence_in_progress) continue;
            goto human_player_reset; // reset move choice
        }
        get_piece_move_pos(board, move_pos, choosen_idx1);
        if (!GET_BEATING_POS_FLAG(move_pos)) break; // check if more beatings in sequence
        beating_sequence_in_progress = true;
    }
    FLIP_TURN_FLAG(board);
}

void random_player(unsigned int board[4], unsigned int move_pos[4])
{
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dist(0, 0);
    unsigned int choosen_idx1, choosen_idx2, dir = 0, dir_idx_upper_bound, dir_idx_counter = 0;
    bool beating_sequence_in_progress = false, tmp;
    unsigned int (*get_dir_idx_ptr)(unsigned int&);

    get_move_possibility(board, move_pos);
    dir_idx_upper_bound = (GET_NUM_OF_MOVES(move_pos)) - 1;
    dist = std::uniform_int_distribution<>(0, dir_idx_upper_bound);
    choosen_idx1 = dist(gen);
    choosen_idx1 = GET_VAL_MOVE_POS(choosen_idx1, move_pos);
    do 
    {
        get_piece_move_pos(board, move_pos, choosen_idx1);
        dir_idx_upper_bound = (GET_NUM_OF_MOVES(move_pos)) - 1;
        dist = std::uniform_int_distribution<>(0, dir_idx_upper_bound);
        choosen_idx2 = dist(gen);
        for (dir = 0, dir_idx_counter = 0; dir_idx_counter <= dir_idx_upper_bound && dir < 4; ++dir)
        {
            switch (dir)
            {
            case 0:
                get_dir_idx_ptr = &get_left_upper_idx;
                break;
            case 1:
                get_dir_idx_ptr = &get_right_upper_idx;
                break;
            case 2:
                get_dir_idx_ptr = &get_left_lower_idx;
                break;
            case 3:
                get_dir_idx_ptr = &get_right_lower_idx;
                break;
            default: return;
            }
            if (dir_idx_counter == choosen_idx2);
            else if ((GET_BEATING_POS_FLAG(move_pos) && GET_PIECE_BEATING_FLAG(dir, move_pos)) || (!GET_BEATING_POS_FLAG(move_pos) && GET_PIECE_DIR_FLAG(dir, move_pos)))
            {
                ++dir_idx_counter;
                continue;
            }
            else continue;
            
            if (GET_BEATING_POS_FLAG(move_pos) && GET_PIECE_BEATING_FLAG(dir, move_pos))
            {
                tmp = IS_KING((GET_VAL_BOARD(choosen_idx1, board))); // for promotion check
                choosen_idx2 = get_dir_idx_ptr(choosen_idx1);
                move_piece(board, choosen_idx1, get_dir_idx_ptr);
                choosen_idx1 = get_dir_idx_ptr(choosen_idx2);
                if (tmp != (IS_KING((GET_VAL_BOARD(choosen_idx1, board)))))
                {
                    FLIP_TURN_FLAG(board);
                    return;
                }
                break;
            }
            else if (!GET_BEATING_POS_FLAG(move_pos) && GET_PIECE_DIR_FLAG(dir, move_pos))
            {
                move_piece(board, choosen_idx1, get_dir_idx_ptr);
                FLIP_TURN_FLAG(board);
                return;
            }
        }
        if (dir == 4) return;
        get_piece_move_pos(board, move_pos, choosen_idx1);
        if (!GET_BEATING_POS_FLAG(move_pos)) break; // check if more beatings in sequence
        beating_sequence_in_progress = true;
    } while (beating_sequence_in_progress);
    FLIP_TURN_FLAG(board);
}

unsigned int simulate_game(unsigned int board[4])
{
    unsigned int move_pos[4];
    bool game_over = false;

    while (!game_over) // main loop
    {
        random_player(board, move_pos);
        get_move_possibility(board, move_pos);
        if (0 == (GET_NUM_OF_MOVES(move_pos))) game_over = true; // end game if noone can move
    }
    get_end_state(board);
    return board[0];
}

unsigned int count_beating_sequences_for_piece(unsigned int board[4], unsigned int cur_tile_idx, unsigned int dir)
{
    unsigned int piece_pos[4], tmp_board[4]{}, possible_moves = 0, dir_tile_idx;
    bool tmp;
    unsigned int (*get_dir_idx_ptr)(unsigned int&);
    
    tmp_board[0] = board[0]; tmp_board[1] = board[1]; tmp_board[2] = board[2]; tmp_board[3] = board[3];
    get_piece_move_pos(tmp_board, piece_pos, cur_tile_idx);
    switch (dir)
    {
    case 0:
        get_dir_idx_ptr = &get_left_upper_idx;
        break;
    case 1:
        get_dir_idx_ptr = &get_right_upper_idx;
        break;
    case 2:
        get_dir_idx_ptr = &get_left_lower_idx;
        break;
    case 3:
        get_dir_idx_ptr = &get_right_lower_idx;
        break;
    default: system("CLS"); std::cout << "ERROR"; system("pause"); exit(EXIT_FAILURE);
    }
    if (GET_BEATING_POS_FLAG(piece_pos) && GET_PIECE_BEATING_FLAG(dir, piece_pos))
    {
        tmp = IS_KING((GET_VAL_BOARD(cur_tile_idx, tmp_board))); // for promotion check
        dir_tile_idx = get_dir_idx_ptr(cur_tile_idx);
        move_piece(tmp_board, cur_tile_idx, get_dir_idx_ptr);
        cur_tile_idx = get_dir_idx_ptr(dir_tile_idx);
        ++possible_moves;
        if (tmp != (IS_KING((GET_VAL_BOARD(cur_tile_idx, tmp_board)))))
        {
            return possible_moves;
        }
        get_piece_move_pos(tmp_board, piece_pos, cur_tile_idx);
        if (GET_BEATING_POS_FLAG(piece_pos)) // check if more beatings in sequence
        {
            possible_moves = 0;
            for (unsigned int dir = 0; dir < 4; ++dir)
                possible_moves += count_beating_sequences_for_piece(tmp_board, cur_tile_idx, dir);
        }
    }
    return possible_moves;
}

void MTS_CPU_player(unsigned int board[4])
{
    //std::random_device rd;
    //std::mt19937 gen(rd());
    //std::uniform_int_distribution<> dist(0, 0);
    unsigned int move_pos[4]{};// , piece_pos[4]{}, tmp_board[4]{}, possible_moves = 0, ** first_layer;
    unsigned int cur_tile_idx;
    //unsigned int (*get_dir_idx_ptr)(unsigned int&);

    get_move_possibility(board, move_pos);
    for (unsigned int i = 0; i < GET_NUM_OF_MOVES(move_pos); ++i)
    {
        cur_tile_idx = GET_VAL_MOVE_POS(i, move_pos);
        //possible_moves += count_beating_sequences_for_piece(board, cur_tile_idx);
    }
    //first_layer = new unsigned int *[GET_NUM_OF_MOVES(move_pos)];
    //for (unsigned int i = 0; i < GET_NUM_OF_MOVES(move_pos); ++i)
    //    first_layer[i] = new unsigned int[4];
    //(first_layer[move_idx] = board
}

////////////////////////////////////////////////////////////////////////////////

void disp_moveable_pieces(unsigned int board[4], unsigned int move_pos[4])
{
    char cords[2]{'-'};
    std::cout << "Possible moves for " << (GET_TURN_FLAG(board) ? "white" : "black") << " - " << (GET_NUM_OF_MOVES(move_pos)) << std::endl;
    std::cout << "Tiles with moveable pieces: ";
    for (unsigned int i = 0; i < GET_NUM_OF_MOVES(move_pos); ++i)
    {
        translate_idx_to_cords((GET_VAL_MOVE_POS(i, move_pos)), cords);
        std::cout << cords[0] << cords[1] << ' ';
    }
    std::cout << std::endl;
}

void disp_possible_dirs(unsigned int board[4], unsigned int move_pos[4], unsigned int& idx)
{
    char cords[2]{'-'};
    translate_idx_to_cords(idx, cords);

    get_piece_move_pos(board, move_pos, idx);
    if (GET_NUM_OF_MOVES(move_pos))
    {
        std::cout << "Moves possible for piece on " << cords[0] << cords[1] << " - " << (GET_NUM_OF_MOVES(move_pos)) << std::endl;
        if (GET_BEATING_POS_FLAG(move_pos)) std::cout << "BEATING POSSIBLE!" << std::endl;
        std::cout << "List of tiles to choose from: ";
        unsigned int (*get_dir_idx_ptr)(unsigned int&);
        for (unsigned int dir = 0; dir < 4; ++dir)
        {
            switch (dir)
            {
            case 0:
                get_dir_idx_ptr = &get_left_upper_idx;
                break;
            case 1:
                get_dir_idx_ptr = &get_right_upper_idx;
                break;
            case 2:
                get_dir_idx_ptr = &get_left_lower_idx;
                break;
            case 3:
                get_dir_idx_ptr = &get_right_lower_idx;
                break;
            default: break;
            }
            translate_idx_to_cords(get_dir_idx_ptr(idx), cords);
            if (GET_BEATING_POS_FLAG(move_pos) && GET_PIECE_BEATING_FLAG(dir, move_pos)) std::cout << cords[0] << cords[1] << ' ';
            else if (!GET_BEATING_POS_FLAG(move_pos) && GET_PIECE_DIR_FLAG(dir, move_pos)) std::cout << cords[0] << cords[1] << ' ';
        }
        std::cout << std::endl;
    }
    else std::cout << "Movement not possible for piece on " << cords[0] << cords[1] << std::endl;
}

void get_cords_from_console(char cords[2])
{
    while (true)
    {
        std::string input = "";
        std::cout << "Please provide coordinates: ";
        std::getline(std::cin, input);
        if (input.size() != 2)
        {
            std::cout << "Incorrect input length!" << std::endl << std::endl;
            continue;
        }
        cords[0] = toupper(input[0]);
        cords[1] = toupper(input[1]);
        if ((cords[0] == 'A' || cords[0] == 'C' || cords[0] == 'E' || cords[0] == 'G') && (cords[1] == '2' || cords[1] == '4' || cords[1] == '6' || cords[1] == '8')) break;
        else if ((cords[0] == 'B' || cords[0] == 'D' || cords[0] == 'F' || cords[0] == 'H') && (cords[1] == '1' || cords[1] == '3' || cords[1] == '5' || cords[1] == '7')) break;
        std::cout << "Incorrect coordinates given!" << std::endl << std::endl;
    }
}

unsigned int translate_cords_to_idx(const char cords[2])
{
    if (cords[1] < '0' || cords[1] > '8') return 32; // out of bounds
    unsigned int cord1 = cords[1] - '1'; // not '0' because we count cords from 1
    switch (cords[0])
    {
    case 'A':
        if (~cord1 & 1) return 32;
        return cord1 << 2;
    case 'B':
        if (cord1 & 1) return 32;
        return cord1 << 2;
    case 'C':
        if (~cord1 & 1) return 32;
        return (cord1 << 2) + 1;
    case 'D':
        if (cord1 & 1) return 32;
        return (cord1 << 2) + 1;
    case 'E':
        if (~cord1 & 1) return 32;
        return (cord1 << 2) + 2;
    case 'F':
        if (cord1 & 1) return 32;
        return (cord1 << 2) + 2;
    case 'G':
        if (~cord1 & 1) return 32;
        return (cord1 << 2) + 3;
    case 'H':
        if (cord1 & 1) return 32;
        return (cord1 << 2) + 3;
    default:
        return 32;
    }
}

void translate_idx_to_cords(unsigned int idx, char cords[2])
{
    if (idx > 31) {
        cords[0] = '-';
        cords[1] = '-';
        return;
    }
    else if (idx < 4) cords[1] = '1';
    else if (idx >= 4 && idx < 8) cords[1] = '2';
    else if (idx >= 8 && idx < 12) cords[1] = '3';
    else if (idx >= 12 && idx < 16) cords[1] = '4';
    else if (idx >= 16 && idx < 20) cords[1] = '5';
    else if (idx >= 20 && idx < 24) cords[1] = '6';
    else if (idx >= 24 && idx < 28) cords[1] = '7';
    else if (idx >= 28 && idx < 32) cords[1] = '8';
    if ((idx & 7) == 0) cords[0] = 'B';
    else if ((idx & 7) == 1) cords[0] = 'D';
    else if ((idx & 7) == 2) cords[0] = 'F';
    else if ((idx & 7) == 3) cords[0] = 'H';
    else if ((idx & 7) == 4) cords[0] = 'A';
    else if ((idx & 7) == 5) cords[0] = 'C';
    else if ((idx & 7) == 6) cords[0] = 'E';
    else if ((idx & 7) == 7) cords[0] = 'G';
}

// saves end state in board[0], 0 - error, 1 - black win, 2 - white win, 3 - draw 
void get_end_state(unsigned int board[4])
{
    unsigned int move_pos[4];
    
    get_move_possibility(board, move_pos);
    for (unsigned int i = 0; i < 32; ++i)
    {
        move_pos[0] = GET_VAL_BOARD(i, board);
        if (IS_PIECE(move_pos[0]))
        {
            if (IS_WHITE(move_pos[0])) board[1] |= 128;
            if (IS_BLACK(move_pos[0])) board[1] |= 8;
        }
    }
    board[0] = 0;
    if (board[1] & 128) board[0] |= 2;
    if (board[1] & 8) board[0] |= 1;
}

void disp_end_state(unsigned int* board)
{
    system("CLS");
    draw_board(board);
    get_end_state(board);
    if (board[0] & 2 && board[0] & 1) std::cout << std::endl << "Game ended in a draw!" << std::endl << std::endl;
    else if (board[0] & 2) std::cout << std::endl << BG_WHITE_FG_BLACK << "White won!" << BG_BLACK_FG_WHITE << std::endl << std::endl;
    else if (board[0] & 1) std::cout << std::endl << "Black won!" << std::endl << std::endl;
    else if (!board[0]) std::cout << std::endl << "Error occured!" << std::endl << std::endl;
}

////////////////////////////////////////////////////////////////////////////////

int main(int argc, char** argv)
{
    unsigned int board[4];

    unsigned short menu_choice = 0;
    bool player_chosen = false;
    void (*white_player)(unsigned int*, unsigned int*);
    void (*black_player)(unsigned int*, unsigned int*);

    std::cout << BG_WHITE_FG_BLACK << BG_BLACK_FG_WHITE;
    system("cls");
    testing_function();
    while (menu_choice != 2) {
        player_chosen = false;
        std::cout << "1. Start Game - Black Always Begins" << std::endl;
        std::cout << "2. Exit" << std::endl;
        std::cout << "Choice: ";
        std::cin >> menu_choice;
        switch (menu_choice)
        {
        case 1:
            while (!player_chosen)
            {
                system("CLS");
                std::cout << "1. Human Player" << std::endl;
                std::cout << "2. Random Player" << std::endl;
                std::cout << BG_WHITE_FG_BLACK << "White" << BG_BLACK_FG_WHITE << " Player Choice: ";
                std::cin >> menu_choice;
                std::cout << std::endl;
                switch (menu_choice)
                {
                case 1:
                    white_player = &human_player;
                    player_chosen = true;
                    break;
                case 2:
                    white_player = &random_player;
                    player_chosen = true;
                    break;
                default:
                    system("CLS");
                    std::cout << "Please provide a valid choice!" << std::endl << std::endl;
                }
            }
            player_chosen = false;
            while (!player_chosen)
            {
                system("CLS");
                std::cout << "1. Human Player" << std::endl;
                std::cout << "2. Random Player" << std::endl;
                std::cout << "Black Player Choice: ";
                std::cin >> menu_choice;
                std::cout << std::endl;
                switch (menu_choice)
                {
                case 1:
                    black_player = &human_player;
                    player_chosen = true;
                    break;
                case 2:
                    black_player = &random_player;
                    player_chosen = true;
                    break;
                default:
                    system("CLS");
                    std::cout << "Please provide a valid choice!" << std::endl << std::endl;
                }
            }
            menu_choice = 1;
            std::cin.ignore();
            init_board(board);
            game_loop(board, white_player, black_player);
            disp_end_state(board);
            system("pause");
            system("CLS");
            break;
        case 2:
            break;
        default:
            system("CLS");
            std::cout << "Please provide a valid choice!" << std::endl << std::endl;
            break;
        }
    }
    exit(EXIT_SUCCESS);
}

////////////////////////////////////////////////////////////////////////////////
void testing_function()
{
    unsigned int board[4];
    unsigned int move_possibility[3]{};

    //init_board(board);
    //draw_board(board);

    //test_get_move_possibility(board, move_possibility);

    //FLIP_TURN_FLAG(board);
    //test_get_move_possibility(board, move_possibility);
    //std::cout << std::endl;

    //std::cout << std::endl;
    ////test_get_idx_funs(board);
    ////std::cout << std::endl;
    //test_translate_cords_to_idx();
    //test_translate_idx_to_cords();
    //std::cout << std::endl;
    ////test_get_move_possibility_init_loop(board);
    ////std::cout << std::endl;
    ////test_get_piece_move_pos(board, move_possibility, 9, 6);

    init_board(board);
    board[0] = 1074020352;
    board[1] = 1178861808;
    board[2] = 102;
    board[3] = 419424;
    FLIP_TURN_FLAG(board);
    system("CLS");
    draw_board(board);
    test_get_move_possibility(board, move_possibility);
    test_get_piece_move_pos(board, move_possibility, 4);
    game_loop(board, &random_player, &random_player);
    disp_end_state(board);
    system("pause");

    //unsigned int game_count = 1000000;
    //std::chrono::steady_clock::time_point start, finish;
    //std::chrono::duration<double> elapsed;
    //
    //start = std::chrono::high_resolution_clock::now();
    //for (unsigned int i = 0; i < game_count; ++i)
    //{
    //    init_board(board);
    //    game_loop(board, &random_player, &random_player);
    //    get_end_state(board);
    //    //disp_end_state(board);
    //}
    //finish = std::chrono::high_resolution_clock::now();
    //elapsed = (finish - start);

    //std::cout << "Games played: " << game_count << std::endl;
    //std::cout << "Elapsed time: " << elapsed.count() << std::endl;
    //std::cout << "Average time: " << elapsed.count() / game_count << std::endl;
    exit(EXIT_SUCCESS);
}

void test_get_idx_funs(unsigned int board[4])
{
    //test top
    unsigned int tmp = 0;
    std::cout << (32 == get_left_upper_idx(tmp)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp);
    std::cout << std::endl << (32 == get_right_upper_idx(tmp)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp);
    std::cout << std::endl << (4 == get_left_lower_idx(tmp)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp);
    std::cout << std::endl << (5 == get_right_lower_idx(tmp)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp);
    std::cout << std::endl;

    tmp = 1;
    std::cout << std::endl << (32 == get_left_upper_idx(tmp)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp);
    std::cout << std::endl << (32 == get_right_upper_idx(tmp)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp);
    std::cout << std::endl << (5 == get_left_lower_idx(tmp)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp);
    std::cout << std::endl << (6 == get_right_lower_idx(tmp)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp);
    std::cout << std::endl;

    tmp = 3;
    std::cout << std::endl << (32 == get_left_upper_idx(tmp)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp);
    std::cout << std::endl << (32 == get_right_upper_idx(tmp)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp);
    std::cout << std::endl << (7 == get_left_lower_idx(tmp)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp);
    std::cout << std::endl << (32 == get_right_lower_idx(tmp)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp);
    std::cout << std::endl;

    // test even
    tmp = 4;
    std::cout << std::endl << (32 == get_left_upper_idx(tmp)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp);
    std::cout << std::endl << (0 == get_right_upper_idx(tmp)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp);
    std::cout << std::endl << (32 == get_left_lower_idx(tmp)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp);
    std::cout << std::endl << (8 == get_right_lower_idx(tmp)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp);
    std::cout << std::endl;

    tmp = 5;
    std::cout << std::endl << (0 == get_left_upper_idx(tmp)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp);
    std::cout << std::endl << (1 == get_right_upper_idx(tmp)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp);
    std::cout << std::endl << (8 == get_left_lower_idx(tmp)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp);
    std::cout << std::endl << (9 == get_right_lower_idx(tmp)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp);
    std::cout << std::endl;

    tmp = 7;
    std::cout << std::endl << (2 == get_left_upper_idx(tmp)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp);
    std::cout << std::endl << (3 == get_right_upper_idx(tmp)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp);
    std::cout << std::endl << (10 == get_left_lower_idx(tmp)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp);
    std::cout << std::endl << (11 == get_right_lower_idx(tmp)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp);
    std::cout << std::endl;

    //test odd
    tmp = 8;
    std::cout << std::endl << (4 == get_left_upper_idx(tmp)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp);
    std::cout << std::endl << (5 == get_right_upper_idx(tmp)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp);
    std::cout << std::endl << (12 == get_left_lower_idx(tmp)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp);
    std::cout << std::endl << (13 == get_right_lower_idx(tmp)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp);
    std::cout << std::endl;

    tmp = 9;
    std::cout << std::endl << (5 == get_left_upper_idx(tmp)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp);
    std::cout << std::endl << (6 == get_right_upper_idx(tmp)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp);
    std::cout << std::endl << (13 == get_left_lower_idx(tmp)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp);
    std::cout << std::endl << (14 == get_right_lower_idx(tmp)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp);
    std::cout << std::endl;

    tmp = 11;
    std::cout << std::endl << (7 == get_left_upper_idx(tmp)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp);
    std::cout << std::endl << (32 == get_right_upper_idx(tmp)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp);
    std::cout << std::endl << (15 == get_left_lower_idx(tmp)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp);
    std::cout << std::endl << (32 == get_right_lower_idx(tmp)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp);
    std::cout << std::endl;

    //test bottom
    tmp = 28;
    std::cout << std::endl << (32 == get_left_upper_idx(tmp)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp);
    std::cout << std::endl << (24 == get_right_upper_idx(tmp)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp);
    std::cout << std::endl << (32 == get_left_lower_idx(tmp)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp);
    std::cout << std::endl << (32 == get_right_lower_idx(tmp)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp);
    std::cout << std::endl;

    tmp = 29;
    std::cout << std::endl << (24 == get_left_upper_idx(tmp)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp);
    std::cout << std::endl << (25 == get_right_upper_idx(tmp)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp);
    std::cout << std::endl << (32 == get_left_lower_idx(tmp)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp);
    std::cout << std::endl << (32 == get_right_lower_idx(tmp)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp);
    std::cout << std::endl;

    tmp = 31;
    std::cout << std::endl << (26 == get_left_upper_idx(tmp)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp);
    std::cout << std::endl << (27 == get_right_upper_idx(tmp)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp);
    std::cout << std::endl << (32 == get_left_lower_idx(tmp)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp);
    std::cout << std::endl << (32 == get_right_lower_idx(tmp)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp);
    std::cout << std::endl;
}

void test_get_move_possibility(unsigned int board[4], unsigned int move_pos[4])
{
    get_move_possibility(board, move_pos);
    std::cout << std::endl << "Possible moves " << (GET_TURN_FLAG(board) ? "for white: " : "for black: ") << (GET_NUM_OF_MOVES(move_pos)) << std::endl;
    std::cout << "Indices of pawns possible to move: ";
    for (unsigned int i = 0; i < GET_NUM_OF_MOVES(move_pos); ++i)
    {
        std::cout << (GET_VAL_MOVE_POS(i, move_pos)) << ' ';
    }
    std::cout << std::endl;
}

void test_get_move_possibility_board_init(unsigned int board[4], unsigned int test_choice)
{
    init_board(board);
    switch (test_choice)
    {
    case 0:
        // black bottom - outdated
        board[0] = 1717986918; //1st 2nd rows
        board[1] = 26214; //3rd 4th rows
        board[2] = 1145307136; //5th 6th rows
        board[3] = 1145324612; //7th 8th rows
        break;
    case 1:
        // test 1 - white forward beating
        // expected - white = 2 moves, idx : 22 23
        // expected - black = 4 moves, idx : 8 9 10 11
        board[2] = 1717986304; //5th 6th rows
        break;
    case 2:
        // test 2 - white no backward beating, black forward beating
        // expected - white = 2 moves, idx: 19 23
        // expected - black = 2 moves, idx: 5 18
        board[1] = 1078198368;
        board[2] = 1717986304;
        board[3] = 1717986822;
        break;
    case 3:
        // test 3 - black no backward beating
        // expected - white = 5 moves, idx: 9 20 21 22 23
        // expected - black = 1 move,  idx: 5
        board[0] = 1078215748;
        board[1] = 1078198368;
        break;
    case 4:
        // test 4
        // expected - white = 5 moves, idx: 9 20 21 22 23
        // expected - black = 8 moves, idx: 0 1 4 6 7 12 13 15
        board[0] = 1141130308;
        board[1] = 1078198368;
        break;
    case 5:
        // test 5 - black King backward beating
        // expected - white = 5 moves, idx: 9 20 21 22 23
        // expected - black = 1 move,  idx: 5 13
        board[0] = 1078215748;
        board[1] = 1079246944;
        break;
    case 6:
        // test 6 - white King backward beating
        // expected - white = 1 move,  idx: 9
        // expected - black = 8 moves, idx: 0 1 4 6 7 12 13 15
        board[0] = 1141130308;
        board[1] = 1078198384;
        break;
    case 7:
        // test 7 - promotion switch turn
        board[0] = 1073759296;
        board[1] = 17412;
        board[2] = 1617168128;
        board[3] = 1711695462;
    default:
        break;
    }
}

void test_get_move_possibility_init_loop(unsigned int board[4], int lower_bound, int upper_bound)
{
    for (int i = lower_bound; i < upper_bound; ++i)
    {
        system("pause");
        test_get_move_possibility_board_init(board, i);
        system("CLS");
        draw_board(board);

        std::cout << "Running test " << i << std::endl;

        unsigned int move_possibility[3]{};
        test_get_move_possibility(board, move_possibility);

        FLIP_TURN_FLAG(board);
        test_get_move_possibility(board, move_possibility);
        std::cout << std::endl;

        std::cout << std::endl;
        test_translate_cords_to_idx();
        std::cout << std::endl;
    }
}

void test_get_piece_move_pos(unsigned int board[4], unsigned int move_pos[4], unsigned int idx)
{
    char cords[2];
    translate_idx_to_cords(idx, cords);

    system("CLS");
    draw_board(board);
    test_translate_cords_to_idx();
    test_translate_idx_to_cords();
    std::cout << std::endl;

    test_get_move_possibility(board, move_pos);

    FLIP_TURN_FLAG(board);
    test_get_move_possibility(board, move_pos);
    std::cout << std::endl;

    get_piece_move_pos(board, move_pos, idx);
    if (GET_NUM_OF_MOVES(move_pos))
    {
        std::cout << "Moves possible for piece on " << cords[0] << cords[1] << " - " << (GET_NUM_OF_MOVES(move_pos)) << std::endl;
        if (GET_BEATING_POS_FLAG(move_pos)) std::cout << "BEATING POSSIBLE!" << std::endl;
        std::cout << "List of tiles to choose from: ";
        unsigned int (*get_dir_idx_ptr)(unsigned int&);
        for (unsigned int dir = 0; dir < 4; ++dir)
        {
            switch (dir)
            {
            case 0:
                get_dir_idx_ptr = &get_left_upper_idx;
                break;
            case 1:
                get_dir_idx_ptr = &get_right_upper_idx;
                break;
            case 2:
                get_dir_idx_ptr = &get_left_lower_idx;
                break;
            case 3:
                get_dir_idx_ptr = &get_right_lower_idx;
                break;
            default: break;
            }
            translate_idx_to_cords(get_dir_idx_ptr(idx), cords);
            if (GET_BEATING_POS_FLAG(move_pos) && GET_PIECE_BEATING_FLAG(dir, move_pos)) std::cout << cords[0] << cords[1] << ' ';
            else if (!GET_BEATING_POS_FLAG(move_pos) && GET_PIECE_DIR_FLAG(dir, move_pos)) std::cout << cords[0] << cords[1] << ' ';
        }
        std::cout << std::endl;
    }
    else std::cout << "Movement not possible for piece on " << cords[0] << cords[1] << std::endl;
}

void test_translate_cords_to_idx()
{
    char cords[2] = {'A', '1'};
    for (char c2 = '1'; c2 < '9'; ++c2)
    {
        cords[1] = c2;
        for (char c1 = 'A'; c1 < 'I'; ++c1)
        {
            cords[0] = c1;
            unsigned int idx = translate_cords_to_idx(cords);
            std::cout << cords[0] << cords[1] << ": " << (32 == idx ? "--" : std::to_string(idx)) << '\t';
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

void test_translate_idx_to_cords()
{
    char cords[2] = { '-', '-' };
    std::cout << '\t';
    for (unsigned int idx = 0; idx < 32; ++idx)
    {
        translate_idx_to_cords(idx, cords);
        std::cout << (idx > 9 ? '\0' : ' ') << idx << ": " << cords[0] << cords[1] << "\t\t";
        if ((idx & 3) == 3) std::cout << std::endl;
        if ((idx & 7) == 7) std::cout << '\t';
    }
    std::cout << std::endl;
}

//void move_piece(unsigned int board[4], unsigned int& cur_tile_idx, unsigned int (*get_dir_idx_ptr)(unsigned int&, unsigned int*))
//{
//    if (cur_tile_idx > 31) return;
//
//    unsigned int other_tile_idx = get_dir_idx_ptr(cur_tile_idx, board);
//    if (other_tile_idx == 32) return;
//
//    unsigned int cur_tile = GET_VAL_BOARD_S(cur_tile_idx, board);
//    unsigned int other_tile = GET_VAL_BOARD_S(other_tile_idx, board);
//    if (!IS_PIECE(other_tile))
//    {
//        SET_VAL_BOARD(other_tile_idx, cur_tile, board);
//        SET_VAL_BOARD(cur_tile_idx, 0, board);
//    }
//    else if (IS_WHITE(cur_tile) == IS_WHITE(other_tile)) return;
//    else
//    {
//        unsigned int other_tile_idx2 = get_dir_idx_ptr(other_tile_idx, board);
//        if (GET_VAL_BOARD_S(other_tile_idx2, board)) return;
//        SET_VAL_BOARD(other_tile_idx2, cur_tile, board);
//        SET_VAL_BOARD(other_tile_idx, 0, board);
//        SET_VAL_BOARD(cur_tile_idx, 0, board);
//    }
//}

//void bench(unsigned int board[4])
//{
//    std::chrono::steady_clock::time_point start, finish, start2, finish2;
//    std::chrono::duration<double> elapsed, elapsed2;
//
//    start = std::chrono::high_resolution_clock::now();
//    for (unsigned int i = 0; i < 1000000; ++i)
//    {
//        for (unsigned int idx = 0; idx < 32; ++idx)
//        {
//            // old - GET_VAL_BOARD_S(idx, board);
//            int tmp = GET_VAL_BOARD_S(idx, board) & 3;
//            //int tmp = GET_VAL_BOARD_S(idx, board) << 2;
//            //int tmp = GET_VAL_BOARD_S(idx, board) >> 2;
//            //int tmp = GET_VAL_BOARD_S(idx, board);
//            //int tmp = 16 | 123;
//        }
//    }
//    finish = std::chrono::high_resolution_clock::now();
//    elapsed = (finish - start) / 1000000;
//
//    start2 = std::chrono::high_resolution_clock::now();
//    for (unsigned int i = 0; i < 1000000; ++i)
//    {
//        for (unsigned int idx = 0; idx < 32; ++idx)
//        {
//            // old - GET_VAL_BOARD_S2(idx, board);
//            int tmp = GET_VAL_BOARD_S(idx, board) % 4;
//            //int tmp = GET_VAL_BOARD_S(idx, board) * 4;
//            //int tmp = GET_VAL_BOARD_S(idx, board) / 4;
//            //int tmp = GET_VAL_BOARD_S(idx, board) / 4;
//            //int tmp = 16 ^ 123;
//        }
//    }
//    finish2 = std::chrono::high_resolution_clock::now();
//    elapsed2 = (finish2 - start2) / 1000000;
//
//    //old - std::cout << "Average time for GET_VAL_BOARD_S:  " << elapsed.count() << std::endl;
//    //old - std::cout << "Average time for GET_VAL_BOARD_S2: " << elapsed2.count() << std::endl << std::endl;
//    std::cout << "Average time for & 3:\t" << elapsed.count() << std::endl;
//    std::cout << "Average time for % 4:\t" << elapsed2.count() << std::endl << std::endl;
//    //std::cout << "Average time for << 2:\t" << elapsed.count() << std::endl;
//    //std::cout << "Average time for * 4:\t" << elapsed2.count() << std::endl << std::endl;
//    //std::cout << "Average time for >> 2:\t" << elapsed.count() << std::endl;
//    //std::cout << "Average time for / 4:\t" << elapsed2.count() << std::endl << std::endl;
//    //std::cout << "Average time for get:\t" << elapsed.count() << std::endl;
//    //std::cout << "Average time for get/4:\t" << elapsed2.count() << std::endl << std::endl;
//    //std::cout << "Average time for | :\t" << elapsed.count() << std::endl;
//    //std::cout << "Average time for ^ :\t" << elapsed2.count() << std::endl << std::endl;
//}