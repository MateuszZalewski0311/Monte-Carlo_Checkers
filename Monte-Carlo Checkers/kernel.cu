// includes, system
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <string>
#include <random>
#include <chrono>
#include <algorithm>

// includes, cuda
#include "hip/hip_runtime.h"
#include ""

////////////////////////////////////////////////////////////////////////////////
#define BG_BBLUE_FG_BLACK "\033[3;104;30m"
#define BG_BLUE_FG_BLACK "\033[3;44;30m"
#define BG_BLUE_FG_WHITE "\033[3;44;37m"
#define BG_BLACK_FG_WHITE "\033[0m"

// 0 - 0000 = empty
// 4 - 0100 = black man
// 5 - 0101 = black king
// 6 - 0110 = white man
// 7 - 0111 = white king
//
// 8 - 1000 = out of bounds

////////////////////////////////////////////////////////////////////////////////
void init_board(unsigned int board[4]);
void draw_board(unsigned int board[4]);
inline unsigned int get_val(unsigned int& idx, unsigned int board[4]);
inline bool is_empty(unsigned int tile);
inline bool is_piece(unsigned int tile);
inline bool is_white(unsigned int tile);
inline bool is_black(unsigned int tile);
inline bool is_king(unsigned int tile);
unsigned int get_left_upper_idx(unsigned int& cur_tile_idx, unsigned int board[4]);
unsigned int get_right_upper_idx(unsigned int& cur_tile_idx, unsigned int board[4]);
unsigned int get_left_lower_idx(unsigned int& cur_tile_idx, unsigned int board[4]);
unsigned int get_right_lower_idx(unsigned int& cur_tile_idx, unsigned int board[4]);
void get_move_possibility_loop_fun(unsigned int board[4], unsigned int move_pos[3], unsigned int& cur_idx, unsigned int& moves_idx, bool& whites_turn);
void get_move_possibility(unsigned int board[4], unsigned int move_pos[3], bool whites_turn);
unsigned int translate_cords_to_idx(const char cords[2]);
////////////////////////////////////////////////////////////////////////////////
void test_get_idx_funs(unsigned int board[4]);
void test_get_move_possibility(unsigned int board[4], unsigned int move_possibility[3], bool whites_turn);
void test_translate_cords_to_idx(unsigned int board[4]);
void bench(unsigned int board[4]);
////////////////////////////////////////////////////////////////////////////////
void init_board(unsigned int board[4])
{
    // black bottom - outdated
    //board[0] = 1717986918; //1st 2nd rows
    //board[1] = 26214; //3rd 4th rows
    //board[2] = 1145307136; //5th 6th rows
    //board[3] = 1145324612; //7th 8th rows

    // white bottom
    board[0] = 1145324612; //1st 2nd rows
    board[1] = 17476; //3rd 4th rows
    board[2] = 1717960704; //5th 6th rows
    board[3] = 1717986918; //7th 8th rows

    // white forward beating move check
    board[2] = 1717986304; //5th 6th rows

    // black forward beating move check
    // board[1] = 4613734; //3rd 4th rows
}

void draw_board(unsigned int board[4])
{
    unsigned short i = 0, left_side_idx = 1;
    bool white_first = true;

    std::cout << BG_BBLUE_FG_BLACK << "   ";
    for (char c = 'A'; c != 'I'; ++c)
        std::cout << ' ' << c << ' ';
    std::cout << BG_BLACK_FG_WHITE << std::endl;

    for (; i < 4; ++i) // i = board_idx
    {
        for (unsigned int j = 0; j < 8; ++j) // j = tile_in_board_idx
        {
            unsigned int tile = board[i] << (28 - (j << 2)) >> 28;
            
            if (j == 0 || j == 4) std::cout << BG_BBLUE_FG_BLACK << ' ' << left_side_idx++ << ' ';

            if (white_first) std::cout << BG_BBLUE_FG_BLACK << "   ";

            if (is_piece(tile))
            {
                if (is_white(tile)) std::cout << BG_BLUE_FG_WHITE;
                else std::cout << BG_BLUE_FG_BLACK;
                if (is_king(tile)) std::cout << " K ";
                else std::cout << " @ ";
            }
            else std::cout << BG_BLUE_FG_BLACK << "   ";
            
            if (!white_first) std::cout << BG_BBLUE_FG_BLACK << "   ";

            if ((j & 3) == 3) // swap colors for second row
            {
                std::cout << BG_BLACK_FG_WHITE << std::endl;
                white_first = !white_first;
            }
        }
    }
    std::cout << BG_BLACK_FG_WHITE << std::endl;
}

inline unsigned int get_val(unsigned int& idx, unsigned int board[4]) 
{
    return idx > 31 ? 8 : board[idx >> 3] << 28 - ((idx & 7) << 2) >> 28;
}

// this is slower
//unsigned int get_val2(unsigned int& idx, unsigned int board[4]) 
//{
//    if (idx > 31) return 8;
//    return (board[idx >> 3] & (15 << ((idx & 7) << 2))) >> ((idx & 7) << 2);
//}

inline bool is_empty(unsigned int tile)
{
    return !tile;
}

inline bool is_piece(unsigned int tile)
{
    return tile & 4;
}

inline bool is_white(unsigned int tile)
{
    return tile & 2;
}

inline bool is_black(unsigned int tile)
{
    return ~tile & 2;
}

inline bool is_king(unsigned int tile)
{
    return tile & 1;
}

////////////////////////////////////////////////////////////////////////////////

unsigned int get_left_upper_idx(unsigned int& cur_tile_idx, unsigned int board[4])
{
    if (cur_tile_idx > 31 || !(cur_tile_idx >> 2)) return 32; // second condition is top row
    if (cur_tile_idx & 4) // even row (counting from 1)
    {
        if (cur_tile_idx & 3) // if not left-most
            return cur_tile_idx - 5;
        return 32;
    }
    else // odd row
    {
        return cur_tile_idx - 4;
    }
}

unsigned int get_right_upper_idx(unsigned int& cur_tile_idx, unsigned int board[4])
{
    if (cur_tile_idx > 31 || !(cur_tile_idx >> 2)) return 32; // second cond chcks if top row
    if (cur_tile_idx & 4) // even row (counting from 1)
    {
        return cur_tile_idx - 4;
    }
    else // odd row
    {
        if (~cur_tile_idx & 3) // if not right-most
            return cur_tile_idx - 3;
        return 32;
    }
}

unsigned int get_left_lower_idx(unsigned int& cur_tile_idx, unsigned int board[4])
{
    if (cur_tile_idx > 31 || (cur_tile_idx >> 2) == 7) return 32; // second cond chcks if bottom row
    if (cur_tile_idx & 4) // even row (counting from 1)
    {
        if (cur_tile_idx & 3) // if not left-most
            return cur_tile_idx + 3;
        return 32;
    }
    else // odd row
    {
        return cur_tile_idx + 4;
    }
}

unsigned int get_right_lower_idx(unsigned int& cur_tile_idx, unsigned int board[4])
{
    if(cur_tile_idx > 31 || (cur_tile_idx >> 2) == 7) return 32; // second cond chcks if bottom row
    if (cur_tile_idx & 4) // even row (counting from 1)
    {
        return cur_tile_idx + 4;
    }
    else // odd row
    {
        if (~cur_tile_idx & 3) // if not right-most
            return cur_tile_idx + 5;
        return 32;
    }
}

void get_move_possibility_loop_fun(unsigned int board[4], unsigned int move_pos[3], unsigned int& cur_idx, unsigned int& moves_idx, bool& whites_turn)
{
    unsigned int tile, tmp_idx, result;
    tile = get_val(cur_idx, board);
    if (is_piece(tile) && (whites_turn == is_white(tile)))
    {
        unsigned int (*get_dir_idx_ptr)(unsigned int&, unsigned int*);
        for (unsigned int direction = 0; direction < 4; ++direction)
        {
            if (whites_turn == (bool)(direction & 2) && !is_king(tile)) // do not check backwards movement
                continue;
            switch (direction)
            {
            case 0:
                get_dir_idx_ptr = &get_left_upper_idx;
                break;
            case 1:
                get_dir_idx_ptr = &get_right_upper_idx;
                break;
            case 2:
                get_dir_idx_ptr = &get_left_lower_idx;
                break;
            case 3:
                get_dir_idx_ptr = &get_right_lower_idx;
                break;
            default: return;
            }
            tmp_idx = get_dir_idx_ptr(cur_idx, board);
            result = get_val(tmp_idx, board);
            if (is_empty(result))
            {
                move_pos[moves_idx >> 2] ^= cur_idx << ((moves_idx & 3) << 3);
                ++moves_idx;
                return;
            }
            else if (whites_turn != is_white(result) && is_piece(result)) // is_piece = out of bounds guard
            {
                tmp_idx = get_dir_idx_ptr(tmp_idx, board);
                result = get_val(tmp_idx, board);
                if (is_empty(result))
                {
                    move_pos[moves_idx >> 2] ^= cur_idx << ((moves_idx & 3) << 3);
                    ++moves_idx;
                    return;
                }
            }
        }
    }
}

void get_move_possibility(unsigned int board[4], unsigned int move_pos[3], bool whites_turn)
{
    move_pos[0] = move_pos[1] = move_pos[2] = 0;
    unsigned int moves_idx = 0;
    for (unsigned int i = 0; i < 32; ++i)
        get_move_possibility_loop_fun(board, move_pos, i, moves_idx, whites_turn);
    move_pos[2] ^= moves_idx << 16; // record number of possible moves
}

unsigned int translate_cords_to_idx(const char cords[2])
{
    if (cords[1] < '0' || cords[1] > '8') return 32; // out of bounds
    unsigned int cord1 = cords[1] - '1'; // not '0' because we count cords from 1
    switch (cords[0])
    {
    case 'A':
        if (~cord1 & 1) return 32;
        return cord1 << 2;
    case 'B':
        if (cord1 & 1) return 32;
        return cord1 << 2;
    case 'C':
        if (~cord1 & 1) return 32;
        return (cord1 << 2) + 1;
    case 'D':
        if (cord1 & 1) return 32;
        return (cord1 << 2) + 1;
    case 'E':
        if (~cord1 & 1) return 32;
        return (cord1 << 2) + 2;
    case 'F':
        if (cord1 & 1) return 32;
        return (cord1 << 2) + 2;
    case 'G':
        if (~cord1 & 1) return 32;
        return (cord1 << 2) + 3;
    case 'H':
        if (cord1 & 1) return 32;
        return (cord1 << 2) + 3;
    default:
        return 32;
    }
}

int main(int argc, char** argv)
{
    unsigned int board[4];

    init_board(board);
    draw_board(board);

    unsigned int move_possibility[3]{};

    bool whites_turn = true;
    test_get_move_possibility(board, move_possibility, whites_turn);
    std::cout << std::endl;

    std::cout << std::endl;
    //test_get_idx_funs(board);
    //std::cout << std::endl;
    test_translate_cords_to_idx(board);
    std::cout << std::endl;

    //bench(board);

    return 0;
}

////////////////////////////////////////////////////////////////////////////////
void test_get_idx_funs(unsigned int board[4])
{
    //test top
    unsigned int tmp = 0;
    std::cout << (32 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (32 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (4 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (5 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    tmp = 1;
    std::cout << std::endl << (32 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (32 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (5 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (6 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    tmp = 3;
    std::cout << std::endl << (32 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (32 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (7 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (32 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    // test even
    tmp = 4;
    std::cout << std::endl << (32 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (0 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (32 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (8 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    tmp = 5;
    std::cout << std::endl << (0 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (1 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (8 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (9 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    tmp = 7;
    std::cout << std::endl << (2 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (3 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (10 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (11 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    //test odd
    tmp = 8;
    std::cout << std::endl << (4 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (5 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (12 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (13 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    tmp = 9;
    std::cout << std::endl << (5 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (6 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (13 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (14 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    tmp = 11;
    std::cout << std::endl << (7 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (32 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (15 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (32 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    //test bottom
    tmp = 28;
    std::cout << std::endl << (32 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (24 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (32 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (32 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    tmp = 29;
    std::cout << std::endl << (24 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (25 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (32 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (32 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    tmp = 31;
    std::cout << std::endl << (26 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (27 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (32 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (32 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;
}

void test_get_move_possibility(unsigned int board[4], unsigned int move_possibility[3], bool whites_turn)
{
    get_move_possibility(board, move_possibility, whites_turn);
    std::cout << std::endl << "Possible moves " << (whites_turn ? "for white: " : "for black: ") << (move_possibility[2] >> 16) << std::endl;
    std::cout << "Indices of pawns possible to move: ";
    for (unsigned int i = 0; i < move_possibility[2] >> 16; ++i)
    {
        std::cout << (move_possibility[i >> 2] << 24 - ((i & 3) << 3) >> 24) << ' ';
    }
    std::cout << std::endl;
}

void test_translate_cords_to_idx(unsigned int board[4])
{
    char cords[2] = {'A', '1'};
    for (char c2 = '1'; c2 < '9'; ++c2)
    {
        cords[1] = c2;
        for (char c1 = 'A'; c1 < 'I'; ++c1)
        {
            cords[0] = c1;
            unsigned int idx = translate_cords_to_idx(cords);
            std::cout << cords[0] << cords[1] << ": " << (32 == idx ? "--" : std::to_string(idx)) << '\t';
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

void bench(unsigned int board[4])
{
    std::chrono::steady_clock::time_point start, finish, start2, finish2;
    std::chrono::duration<double> elapsed, elapsed2;

    start = std::chrono::high_resolution_clock::now();
    for (unsigned int i = 0; i < 1000000; ++i)
    {
        for (unsigned int idx = 0; idx < 32; ++idx)
        {
            //get_val(idx, board);
            int tmp = get_val(idx, board) & 3;
            //int tmp = get_val(idx, board) << 2;
            //int tmp = get_val(idx, board) >> 2;
            //int tmp = get_val(idx, board);
        }
    }
    finish = std::chrono::high_resolution_clock::now();
    elapsed = (finish - start) / 1000000;

    start2 = std::chrono::high_resolution_clock::now();
    for (unsigned int i = 0; i < 1000000; ++i)
    {
        for (unsigned int idx = 0; idx < 32; ++idx)
        {
            //get_val2(idx, board);
            int tmp = get_val(idx, board) % 4;
            //int tmp = get_val(idx, board) * 4;
            //int tmp = get_val(idx, board) / 4;
            //int tmp = get_val(idx, board) / 4;
        }
    }
    finish2 = std::chrono::high_resolution_clock::now();
    elapsed2 = (finish2 - start2) / 1000000;

    //std::cout << "Average time for get_val:  " << elapsed.count() << std::endl;
    //std::cout << "Average time for get_val2: " << elapsed2.count() << std::endl << std::endl;
    std::cout << "Average time for & 3:\t" << elapsed.count() << std::endl;
    std::cout << "Average time for % 4:\t" << elapsed2.count() << std::endl << std::endl;
    //std::cout << "Average time for << 2:\t" << elapsed.count() << std::endl;
    //std::cout << "Average time for * 4:\t" << elapsed2.count() << std::endl << std::endl;
    //std::cout << "Average time for >> 2:\t" << elapsed.count() << std::endl;
    //std::cout << "Average time for / 4:\t" << elapsed2.count() << std::endl << std::endl;
    //std::cout << "Average time for get:\t" << elapsed.count() << std::endl;
    //std::cout << "Average time for get/4:\t" << elapsed2.count() << std::endl << std::endl;
}