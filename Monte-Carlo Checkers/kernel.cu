// includes, system
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <string>
#include <random>
#include <chrono>
#include <algorithm>

// includes, cuda
#include "hip/hip_runtime.h"
#include ""

////////////////////////////////////////////////////////////////////////////////
#define BG_BBLUE_FG_BLACK "\033[3;104;30m"
#define BG_BLUE_FG_BLACK "\033[3;44;30m"
#define BG_BLUE_FG_WHITE "\033[3;44;37m"
#define BG_BLACK_FG_WHITE "\033[0m"

// 0 - 0000 = empty
// 4 - 0100 = black man
// 5 - 0101 = black king
// 6 - 0110 = white man
// 7 - 0111 = white king
//
// 8 - 1000 = out of bounds

////////////////////////////////////////////////////////////////////////////////
void init_board(unsigned int board[4]);
void draw_board(unsigned int board[4]);
inline unsigned int get_val(unsigned int& idx, unsigned int board[4]);
inline bool is_empty(unsigned int& tile);
inline bool is_piece(unsigned int& tile);
inline bool is_white(unsigned int& tile);
inline bool is_black(unsigned int& tile);
inline bool is_king (unsigned int& tile);
unsigned int get_left_upper_idx(unsigned int& cur_tile_idx, unsigned int board[4]);
unsigned int get_right_upper_idx(unsigned int& cur_tile_idx, unsigned int board[4]);
unsigned int get_left_lower_idx(unsigned int& cur_tile_idx, unsigned int board[4]);
unsigned int get_right_lower_idx(unsigned int& cur_tile_idx, unsigned int board[4]);
inline bool get_beating_pos(unsigned int& move_pos);
inline void set_beating_pos(unsigned int& move_pos);
inline bool get_move_check_guard(unsigned int& move_pos);
inline void set_move_check_guard(unsigned int& move_pos);
inline void clear_move_check_guard(unsigned int& move_pos);
inline unsigned int get_num_of_moves(unsigned int& move_pos);
inline void set_num_of_moves(unsigned int& move_pos, unsigned int& num_of_moves);
void get_move_possibility_loop_fun(unsigned int board[4], unsigned int move_pos[3], unsigned int& cur_idx, unsigned int& moves_idx, bool& whites_turn);
void get_move_possibility(unsigned int board[4], unsigned int move_pos[3], bool whites_turn);
////////////////////////////////////////////////////////////////////////////////
unsigned int translate_cords_to_idx(const char cords[2]);
////////////////////////////////////////////////////////////////////////////////
void test_get_idx_funs(unsigned int board[4]);
void test_get_move_possibility(unsigned int board[4], unsigned int move_possibility[3], bool whites_turn);
void test_get_move_possibility_board_init(unsigned int board[4], unsigned int test_choice);
void test_get_move_possibility_init_loop(unsigned int board[4], int lower_bound = 1, int upper_bound = 7);
void test_translate_cords_to_idx();
void test_translate_idx_to_cords();
void bench(unsigned int board[4]);
////////////////////////////////////////////////////////////////////////////////
void init_board(unsigned int board[4])
{
    // white bottom
    board[0] = 1145324612; //1st 2nd rows
    board[1] = 17476; //3rd 4th rows
    board[2] = 1717960704; //5th 6th rows
    board[3] = 1717986918; //7th 8th rows
}

void draw_board(unsigned int board[4])
{
    unsigned short i = 0, left_side_idx = 1;
    bool white_first = true;

    std::cout << BG_BBLUE_FG_BLACK << "   ";
    for (char c = 'A'; c != 'I'; ++c)
        std::cout << ' ' << c << ' ';
    std::cout << BG_BLACK_FG_WHITE << std::endl;

    for (; i < 4; ++i) // i = board_idx
    {
        for (unsigned int j = 0; j < 8; ++j) // j = tile_in_board_idx
        {
            unsigned int tile = board[i] << (28 - (j << 2)) >> 28;
            
            if (j == 0 || j == 4) std::cout << BG_BBLUE_FG_BLACK << ' ' << left_side_idx++ << ' ';

            if (white_first) std::cout << BG_BBLUE_FG_BLACK << "   ";

            if (is_piece(tile))
            {
                if (is_white(tile)) std::cout << BG_BLUE_FG_WHITE;
                else std::cout << BG_BLUE_FG_BLACK;
                if (is_king(tile)) std::cout << " K ";
                else std::cout << " @ ";
            }
            else std::cout << BG_BLUE_FG_BLACK << "   ";
            
            if (!white_first) std::cout << BG_BBLUE_FG_BLACK << "   ";

            if ((j & 3) == 3) // swap colors for second row
            {
                std::cout << BG_BLACK_FG_WHITE << std::endl;
                white_first = !white_first;
            }
        }
    }
    std::cout << BG_BLACK_FG_WHITE << std::endl;
}

inline unsigned int get_val(unsigned int& idx, unsigned int board[4]) 
{
    return idx > 31 ? 8 : board[idx >> 3] << 28 - ((idx & 7) << 2) >> 28;
}

////////////////////////////////////////////////////////////////////////////////

inline bool is_empty(unsigned int& tile)
{
    return !tile;
}

inline bool is_piece(unsigned int& tile)
{
    return tile & 4;
}

inline bool is_white(unsigned int& tile)
{
    return tile & 2;
}

inline bool is_black(unsigned int& tile)
{
    return ~tile & 2;
}

inline bool is_king(unsigned int& tile)
{
    return tile & 1;
}

////////////////////////////////////////////////////////////////////////////////

unsigned int get_left_upper_idx(unsigned int& cur_tile_idx, unsigned int board[4])
{
    if (cur_tile_idx > 31 || !(cur_tile_idx >> 2)) return 32; // second condition is top row
    if (cur_tile_idx & 4) // even row (counting from 1)
    {
        if (cur_tile_idx & 3) // if not left-most
            return cur_tile_idx - 5;
        return 32;
    }
    else // odd row
    {
        return cur_tile_idx - 4;
    }
}

unsigned int get_right_upper_idx(unsigned int& cur_tile_idx, unsigned int board[4])
{
    if (cur_tile_idx > 31 || !(cur_tile_idx >> 2)) return 32; // second cond chcks if top row
    if (cur_tile_idx & 4) // even row (counting from 1)
    {
        return cur_tile_idx - 4;
    }
    else // odd row
    {
        if (~cur_tile_idx & 3) // if not right-most
            return cur_tile_idx - 3;
        return 32;
    }
}

unsigned int get_left_lower_idx(unsigned int& cur_tile_idx, unsigned int board[4])
{
    if (cur_tile_idx > 31 || (cur_tile_idx >> 2) == 7) return 32; // second cond chcks if bottom row
    if (cur_tile_idx & 4) // even row (counting from 1)
    {
        if (cur_tile_idx & 3) // if not left-most
            return cur_tile_idx + 3;
        return 32;
    }
    else // odd row
    {
        return cur_tile_idx + 4;
    }
}

unsigned int get_right_lower_idx(unsigned int& cur_tile_idx, unsigned int board[4])
{
    if(cur_tile_idx > 31 || (cur_tile_idx >> 2) == 7) return 32; // second cond chcks if bottom row
    if (cur_tile_idx & 4) // even row (counting from 1)
    {
        return cur_tile_idx + 4;
    }
    else // odd row
    {
        if (~cur_tile_idx & 3) // if not right-most
            return cur_tile_idx + 5;
        return 32;
    }
}

inline bool get_beating_pos(unsigned int& move_pos)
{
    return (move_pos >> 16) & 1;
}

inline void set_beating_pos(unsigned int& move_pos)
{
    move_pos |= 1 << 16;
}

inline bool get_move_check_guard(unsigned int& move_pos)
{
    return (move_pos >> 17) & 1;
}

inline void set_move_check_guard(unsigned int& move_pos)
{
    move_pos |= 1 << 17;
}

inline void clear_move_check_guard(unsigned int& move_pos)
{
    move_pos &= 4294836223;
}

inline unsigned int get_num_of_moves(unsigned int& move_pos)
{
    return move_pos >> 20;
}

inline void set_num_of_moves(unsigned int& move_pos, unsigned int& num_of_moves)
{
    move_pos |= num_of_moves << 20;
}

void get_move_possibility_loop_fun(unsigned int board[4], unsigned int move_pos[3], unsigned int& cur_idx, unsigned int& moves_idx, bool& whites_turn)
{
    unsigned int tile, tmp_idx, result;
    tile = get_val(cur_idx, board);
    if (is_piece(tile) && (whites_turn == is_white(tile)))
    {
        unsigned int (*get_dir_idx_ptr)(unsigned int&, unsigned int*);
        for (unsigned int direction = 0; direction < 4; ++direction)
        {
            if (whites_turn == (bool)(direction & 2) && !is_king(tile)) // do not check backwards movement
                continue;
            switch (direction)
            {
            case 0:
                get_dir_idx_ptr = &get_left_upper_idx;
                break;
            case 1:
                get_dir_idx_ptr = &get_right_upper_idx;
                break;
            case 2:
                get_dir_idx_ptr = &get_left_lower_idx;
                break;
            case 3:
                get_dir_idx_ptr = &get_right_lower_idx;
                break;
            default: return;
            }
            tmp_idx = get_dir_idx_ptr(cur_idx, board);
            result = get_val(tmp_idx, board);
            if (whites_turn != is_white(result) && is_piece(result)) // is_piece = out of bounds guard
            {
                tmp_idx = get_dir_idx_ptr(tmp_idx, board);
                result = get_val(tmp_idx, board);
                if (is_empty(result))
                {
                    if (!get_beating_pos(move_pos[2])) 
                    {
                        moves_idx = 0;
                        move_pos[0] = move_pos[1] = move_pos[2] = 0;
                        set_beating_pos(move_pos[2]);
                    }
                    move_pos[moves_idx >> 2] |= cur_idx << ((moves_idx & 3) << 3);
                    ++moves_idx;
                    clear_move_check_guard(move_pos[2]);
                    return;
                }
            }
            else if (is_empty(result) && !get_beating_pos(move_pos[2]) && !get_move_check_guard(move_pos[2]))
            {
                move_pos[moves_idx >> 2] |= cur_idx << ((moves_idx & 3) << 3);
                ++moves_idx;
                set_move_check_guard(move_pos[2]);
                continue;
            }
        }
        clear_move_check_guard(move_pos[2]);
    }
}

// Index of tile that can be moved is stored similarly as board representation, but in 8 bits instead of 4 bits
// Additionally some space in move_pos[2] is used for flags and saving number of indexes in the whole array
void get_move_possibility(unsigned int board[4], unsigned int move_pos[3], bool whites_turn)
{
    move_pos[0] = move_pos[1] = move_pos[2] = 0;
    unsigned int moves_idx = 0;
    for (unsigned int i = 0; i < 32; ++i)
        get_move_possibility_loop_fun(board, move_pos, i, moves_idx, whites_turn);
    set_num_of_moves(move_pos[2], moves_idx); // record number of possible moves
}

////////////////////////////////////////////////////////////////////////////////

unsigned int translate_cords_to_idx(const char cords[2])
{
    if (cords[1] < '0' || cords[1] > '8') return 32; // out of bounds
    unsigned int cord1 = cords[1] - '1'; // not '0' because we count cords from 1
    switch (cords[0])
    {
    case 'A':
        if (~cord1 & 1) return 32;
        return cord1 << 2;
    case 'B':
        if (cord1 & 1) return 32;
        return cord1 << 2;
    case 'C':
        if (~cord1 & 1) return 32;
        return (cord1 << 2) + 1;
    case 'D':
        if (cord1 & 1) return 32;
        return (cord1 << 2) + 1;
    case 'E':
        if (~cord1 & 1) return 32;
        return (cord1 << 2) + 2;
    case 'F':
        if (cord1 & 1) return 32;
        return (cord1 << 2) + 2;
    case 'G':
        if (~cord1 & 1) return 32;
        return (cord1 << 2) + 3;
    case 'H':
        if (cord1 & 1) return 32;
        return (cord1 << 2) + 3;
    default:
        return 32;
    }
}

void translate_idx_to_cords(unsigned int idx, char cords[2])
{
    if (idx > 31) {
        cords[0] = '-';
        cords[1] = '-';
        return;
    }
    else if (idx < 4) cords[1] = '1';
    else if (idx >= 4 && idx < 8) cords[1] = '2';
    else if (idx >= 8 && idx < 12) cords[1] = '3';
    else if (idx >= 12 && idx < 16) cords[1] = '4';
    else if (idx >= 16 && idx < 20) cords[1] = '5';
    else if (idx >= 20 && idx < 24) cords[1] = '6';
    else if (idx >= 24 && idx < 28) cords[1] = '7';
    else if (idx >= 28 && idx < 32) cords[1] = '8';
    if ((idx & 7) == 0) cords[0] = 'B';
    else if ((idx & 7) == 1) cords[0] = 'D';
    else if ((idx & 7) == 2) cords[0] = 'F';
    else if ((idx & 7) == 3) cords[0] = 'H';
    else if ((idx & 7) == 4) cords[0] = 'A';
    else if ((idx & 7) == 5) cords[0] = 'C';
    else if ((idx & 7) == 6) cords[0] = 'E';
    else if ((idx & 7) == 7) cords[0] = 'G';
}

////////////////////////////////////////////////////////////////////////////////

int main(int argc, char** argv)
{
    unsigned int board[4];

    init_board(board);
    draw_board(board);

    unsigned int move_possibility[3]{};

    bool whites_turn = true;
    test_get_move_possibility(board, move_possibility, whites_turn);

    whites_turn = false;
    test_get_move_possibility(board, move_possibility, whites_turn);
    std::cout << std::endl;

    std::cout << std::endl;
    //test_get_idx_funs(board);
    //std::cout << std::endl;
    test_translate_cords_to_idx();
    test_translate_idx_to_cords();
    std::cout << std::endl;
    //test_get_move_possibility_init_loop(board);

    //bench(board);

    return 0;
}

////////////////////////////////////////////////////////////////////////////////
void test_get_idx_funs(unsigned int board[4])
{
    //test top
    unsigned int tmp = 0;
    std::cout << (32 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (32 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (4 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (5 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    tmp = 1;
    std::cout << std::endl << (32 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (32 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (5 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (6 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    tmp = 3;
    std::cout << std::endl << (32 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (32 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (7 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (32 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    // test even
    tmp = 4;
    std::cout << std::endl << (32 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (0 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (32 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (8 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    tmp = 5;
    std::cout << std::endl << (0 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (1 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (8 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (9 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    tmp = 7;
    std::cout << std::endl << (2 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (3 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (10 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (11 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    //test odd
    tmp = 8;
    std::cout << std::endl << (4 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (5 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (12 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (13 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    tmp = 9;
    std::cout << std::endl << (5 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (6 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (13 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (14 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    tmp = 11;
    std::cout << std::endl << (7 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (32 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (15 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (32 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    //test bottom
    tmp = 28;
    std::cout << std::endl << (32 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (24 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (32 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (32 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    tmp = 29;
    std::cout << std::endl << (24 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (25 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (32 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (32 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;

    tmp = 31;
    std::cout << std::endl << (26 == get_left_upper_idx(tmp, board)) << ": " << "Left upper to " << tmp << ": " << get_left_upper_idx(tmp, board);
    std::cout << std::endl << (27 == get_right_upper_idx(tmp, board)) << ": " << "Right upper to " << tmp << ": " << get_right_upper_idx(tmp, board);
    std::cout << std::endl << (32 == get_left_lower_idx(tmp, board)) << ": " << "Left lower to " << tmp << ": " << get_left_lower_idx(tmp, board);
    std::cout << std::endl << (32 == get_right_lower_idx(tmp, board)) << ": " << "Right lower to " << tmp << ": " << get_right_lower_idx(tmp, board);
    std::cout << std::endl;
}

void test_get_move_possibility(unsigned int board[4], unsigned int move_possibility[3], bool whites_turn)
{
    get_move_possibility(board, move_possibility, whites_turn);
    std::cout << std::endl << "Possible moves " << (whites_turn ? "for white: " : "for black: ") << get_num_of_moves(move_possibility[2]) << std::endl;
    std::cout << "Indices of pawns possible to move: ";
    for (unsigned int i = 0; i < get_num_of_moves(move_possibility[2]); ++i)
    {
        std::cout << (move_possibility[i >> 2] << 24 - ((i & 3) << 3) >> 24) << ' ';
    }
    std::cout << std::endl;
}

void test_get_move_possibility_board_init(unsigned int board[4], unsigned int test_choice)
{
    init_board(board);
    switch (test_choice)
    {
    case 0:
        // black bottom - outdated
        board[0] = 1717986918; //1st 2nd rows
        board[1] = 26214; //3rd 4th rows
        board[2] = 1145307136; //5th 6th rows
        board[3] = 1145324612; //7th 8th rows
        break;
    case 1:
        // test 1 - white forward beating
        // expected - white = 2 moves, idx : 22 23
        // expected - black = 4 moves, idx : 8 9 10 11
        board[2] = 1717986304; //5th 6th rows
        break;
    case 2:
        // test 2 - white no backward beating, black forward beating
        // expected - white = 2 moves, idx: 19 23
        // expected - black = 2 moves, idx: 5 18
        board[1] = 1078198368;
        board[2] = 1717986304;
        board[3] = 1717986822;
        break;
    case 3:
        // test 3 - black no backward beating
        // expected - white = 5 moves, idx: 9 20 21 22 23
        // expected - black = 1 move,  idx: 5
        board[0] = 1078215748;
        board[1] = 1078198368;
        break;
    case 4:
        // test 4
        // expected - white = 5 moves, idx: 9 20 21 22 23
        // expected - black = 8 moves, idx: 0 1 4 6 7 12 13 15
        board[0] = 1141130308;
        board[1] = 1078198368;
        break;
    case 5:
        // test 5 - black King backward beating
        // expected - white = 5 moves, idx: 9 20 21 22 23
        // expected - black = 1 move,  idx: 5 13
        board[0] = 1078215748;
        board[1] = 1079246944;
        break;
    case 6:
        // test 6 - white King backward beating
        // expected - white = 1 move,  idx: 9
        // expected - black = 8 moves, idx: 0 1 4 6 7 12 13 15
        board[0] = 1141130308;
        board[1] = 1078198384;
        break;
    default:
        break;
    }
}

void test_get_move_possibility_init_loop(unsigned int board[4], int lower_bound, int upper_bound)
{
    for (int i = lower_bound; i < upper_bound; ++i)
    {
        system("pause");
        test_get_move_possibility_board_init(board, i);
        system("CLS");
        draw_board(board);

        std::cout << "Running test " << i << std::endl;

        unsigned int move_possibility[3]{};
        bool whites_turn = true;
        test_get_move_possibility(board, move_possibility, whites_turn);

        whites_turn = false;
        test_get_move_possibility(board, move_possibility, whites_turn);
        std::cout << std::endl;

        std::cout << std::endl;
        //test_get_idx_funs(board);
        //std::cout << std::endl;
        test_translate_cords_to_idx();
        std::cout << std::endl;
    }
}

void test_translate_cords_to_idx()
{
    char cords[2] = {'A', '1'};
    for (char c2 = '1'; c2 < '9'; ++c2)
    {
        cords[1] = c2;
        for (char c1 = 'A'; c1 < 'I'; ++c1)
        {
            cords[0] = c1;
            unsigned int idx = translate_cords_to_idx(cords);
            std::cout << cords[0] << cords[1] << ": " << (32 == idx ? "--" : std::to_string(idx)) << '\t';
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

void test_translate_idx_to_cords()
{
    char cords[2] = { '-', '-' };
    std::cout << '\t';
    for (unsigned int idx = 0; idx < 32; ++idx)
    {
        translate_idx_to_cords(idx, cords);
        std::cout << (idx > 9 ? '\0' : ' ') << idx << ": " << cords[0] << cords[1] << "\t\t";
        if ((idx & 3) == 3) std::cout << std::endl;
        if ((idx & 7) == 7) std::cout << '\t';
    }
    std::cout << std::endl;
}

void bench(unsigned int board[4])
{
    std::chrono::steady_clock::time_point start, finish, start2, finish2;
    std::chrono::duration<double> elapsed, elapsed2;

    start = std::chrono::high_resolution_clock::now();
    for (unsigned int i = 0; i < 1000000; ++i)
    {
        for (unsigned int idx = 0; idx < 32; ++idx)
        {
            //get_val(idx, board);
            int tmp = get_val(idx, board) & 3;
            //int tmp = get_val(idx, board) << 2;
            //int tmp = get_val(idx, board) >> 2;
            //int tmp = get_val(idx, board);
            //int tmp = 16 | 123;
        }
    }
    finish = std::chrono::high_resolution_clock::now();
    elapsed = (finish - start) / 1000000;

    start2 = std::chrono::high_resolution_clock::now();
    for (unsigned int i = 0; i < 1000000; ++i)
    {
        for (unsigned int idx = 0; idx < 32; ++idx)
        {
            //get_val2(idx, board);
            int tmp = get_val(idx, board) % 4;
            //int tmp = get_val(idx, board) * 4;
            //int tmp = get_val(idx, board) / 4;
            //int tmp = get_val(idx, board) / 4;
            //int tmp = 16 ^ 123;
        }
    }
    finish2 = std::chrono::high_resolution_clock::now();
    elapsed2 = (finish2 - start2) / 1000000;

    //std::cout << "Average time for get_val:  " << elapsed.count() << std::endl;
    //std::cout << "Average time for get_val2: " << elapsed2.count() << std::endl << std::endl;
    std::cout << "Average time for & 3:\t" << elapsed.count() << std::endl;
    std::cout << "Average time for % 4:\t" << elapsed2.count() << std::endl << std::endl;
    //std::cout << "Average time for << 2:\t" << elapsed.count() << std::endl;
    //std::cout << "Average time for * 4:\t" << elapsed2.count() << std::endl << std::endl;
    //std::cout << "Average time for >> 2:\t" << elapsed.count() << std::endl;
    //std::cout << "Average time for / 4:\t" << elapsed2.count() << std::endl << std::endl;
    //std::cout << "Average time for get:\t" << elapsed.count() << std::endl;
    //std::cout << "Average time for get/4:\t" << elapsed2.count() << std::endl << std::endl;
    //std::cout << "Average time for | :\t" << elapsed.count() << std::endl;
    //std::cout << "Average time for ^ :\t" << elapsed2.count() << std::endl << std::endl;
}